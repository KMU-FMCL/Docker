
#include <hip/hip_runtime.h>
extern "C" __global__ void vectorAdd(const float *A, const float *B, float *C, int N)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx < N)
    C[idx] = A[idx] + B[idx];
}
